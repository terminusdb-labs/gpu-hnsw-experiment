#include "hip/hip_runtime.h"
#include <stdint.h>
#include <hip/hip_bf16.h>

struct ring_queue_t {
  long *ids; 
  __hip_bfloat16 *priorities;
  size_t head;
  size_t tail;
  size_t capacity;
};

struct pair_t {
  long *id;
  __hip_bfloat16 *priority;
};

namespace ring { 
  __device__ size_t ring_relative_to_absolute(ring_queue_t *rq, size_t relative_index) {
    if (relative_index < (rq->capacity - rq->head)) {
      return rq->head + relative_index;
    }else{
      return relative_index - (rq->capacity - rq->head);
    }
  }

  __device__ size_t ring_absolute_to_relative(ring_queue_t *rq, size_t absolute_index) {
    if (absolute_index < rq->head) {
      return absolute_index + (rq->capacity - rq->head);
    }else{
      return absolute_index - rq->head;
    }
  }

  __device__ bool ring_is_empty(ring_queue_t *rq) {
    if (rq->head == rq->tail) {
      return true;
    } else {
      return false;
    }
  }

  __device__ pair_t ring_first(ring_queue_t *rq) {
    struct pair_t pair;
    if (ring_is_empty(rq)) {
      pair.id = nullptr;
      pair.priority = nullptr;
    } else {
      pair.id = &(rq->ids[rq->head]);
      pair.priority = &(rq->priorities[rq->head]);
    }
    return pair;
  }

  __device__ int ring_advance(ring_queue_t * rq) {
    if (ring_is_empty(rq)) {
      return -1;
    } else {
      if (rq->head == rq->capacity-1) {
	rq->head = 0;
	return 0;
      }else{
	rq->head++;
	return 0;
      }
    }
  }

  __device__ pair_t ring_pop_first(ring_queue_t *rq) {
    struct pair_t pair;
    if (ring_is_empty(rq)) {
      pair.id = nullptr;
      pair.priority = nullptr;
    } else {
      pair.id = &(rq->ids[rq->head]);
      pair.priority = &(rq->priorities[rq->head]);
      ring_advance(rq);
    }
    return pair;
  }

  __device__ size_t ring_len(ring_queue_t *rq) {
    if (rq->tail > rq->head) {
      return (rq->tail - rq->head);
    } else {
      return ((rq->capacity - rq->head) + rq->tail);
    }
  }

  __device__ pair_t ring_get(ring_queue_t *rq, size_t idx) {
    struct pair_t pair;
    if (idx < ring_len(rq)) {
      size_t aidx = ring_relative_to_absolute(rq, idx);
      pair.id = &(rq->ids[aidx]);
      pair.priority = &(rq->priorities[aidx]);
    } else {
      pair.id = nullptr;
      pair.priority = nullptr;
    }
    return pair;
  }

  __device__ int ring_set(ring_queue_t *rq, size_t idx, long id, __hip_bfloat16 priority) {
    size_t aidx = ring_relative_to_absolute(rq, idx);
    rq->ids[aidx] = id;
    rq->priorities[aidx] = priority;
    return 0;
  }

}

extern "C" __global__ size_t ring_relative_to_absolute(ring_queue_t *rq, size_t relative_index) {
  return ring::ring_relative_to_absolute(rq, relative_index);
}

extern "C" __global__ size_t ring_absolute_to_relative(ring_queue_t *rq, size_t absolute_index) {
  return ring::ring_absolute_to_relative(rq, absolute_index);
}

extern "C" __global__ bool ring_is_empty(ring_queue_t *rq) {
  return ring::ring_is_empty(rq);
}

extern "C" __global__ pair_t ring_first(ring_queue_t *rq) {
  return ring::ring_first(rq);
}

extern "C" __global__ int ring_advance(ring_queue_t *rq) {
  return ring::ring_advance(rq);
}

extern "C" __global__ pair_t ring_pop_first(ring_queue_t *rq) {
  return ring::ring_pop_first(rq);
}

extern "C" __global__ size_t ring_len(ring_queue_t *rq) {
  return ring::ring_len(rq);
}

extern "C" __global__ pair_t ring_get(ring_queue_t *rq, size_t idx) {
  return ring::ring_get(rq, idx);
}

extern "C" __global__ int ring_set(ring_queue_t *rq, size_t idx, long id, __hip_bfloat16 priority) {
  return ring::ring_set(rq, idx, id, priority);
}
